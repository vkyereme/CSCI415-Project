
#include <hip/hip_runtime.h>
#include<iostream>
#include<string>
#include<cstring>
#include<ctime>
#include<cstdlib>
#include<sys/time.h>
#include<stdio.h>
#include<iomanip>
/* we need these includes for CUDA's random number stuff */
#include<hiprand/hiprand.h>
#include<hiprand/hiprand_kernel.h>

using namespace std;

#define MAX 26

//int a[1000]; //array of all possible password characters
int b[1000]; //array of attempted password cracks
unsigned long long tries = 0;
char alphabet[] = { 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z' };
size_t result = 1000 * sizeof(float);

int *a = (int *) malloc(result);

void serial_passwordCrack(int length){
bool cracked = false;
do{
    b[0]++;
    for(int i =0; i<length; i++){
        if (b[i] >= 26 + alphabet[i]){ 
            b[i] -= 26; 
            b[i+1]++;
        }else break;
    }
    cracked=true;
    for(int k=0; k<length; k++)
        if(b[k]!=a[k]){
            cracked=false;
            break;
        }
    if( (tries & 0x7ffffff) == 0 )
        cout << "\r       \r   ";
    else if( (tries & 0x1ffffff) == 0 )
        cout << ".";
    tries++;
}while(cracked==false);

}


__global__ void parallel_passwordCrack(int length,int*d_output,int *a)
{	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	bool cracked = false;
        char alphabetTable[] = { 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z' };        
	int newB[1000]; 
 

__shared__ int nIter;
__shared__ int idT;
__shared__ long totalAttempt;

do{

   if(idx == 0){
	nIter = 0;
	totalAttempt = 0;
   }   

   newB[0]++;
    for(int i =0; i<length; i++){
        if (newB[i] >= 26 + alphabetTable[i]){ 
            newB[i] -= 26; 
            newB[i+1]++;
        }else break;
    }
    
    cracked=true;

    for(int k=0; k<length; k++)
    {
        if(newB[k]!=a[k]){
            cracked=false;
            break;
        }else
        {
            cracked = true;
       
        }
    }
    if(cracked && nIter == 0){
      
      idT = idx;
      break;
    }
    else if(nIter){

	break;
    }

    totalAttempt++;
}while(!cracked || !nIter);

if(idx == idT){
        for(int i = 0; i< length; i++){
  
             d_output[i] = newB[i];
    }

 }



}

long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, std::string name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
        std::cout << std::setprecision(5);	
	std::cout << name << ": " << ((float) (end_time - start_time)) / (1000 * 1000) << " sec\n";
	return end_time - start_time;
}



int main()
{
int length; //length of password
int random; //random password to be generated
int *d_input = (int *) malloc(result);

cout << "Enter a password length: ";
cin >> length;
int *h_gpu_result = (int*)malloc(1000*sizeof(int));

srand(time(NULL));

//generating random password
cout << "Random generated password: " << endl;
for (int i =0; i<length; i++){
    
        random = alphabet[(rand()%26)]; 
    
    a[i] = random; //adding random password to array
    cout << char(a[i]);
}cout << "\n" << endl;

long long serial_start_time = start_timer();

 cout << "Serial Password Cracked: " << endl;
 serial_passwordCrack(length);
 cout << "\n";

long long serial_end_time = stop_timer(serial_start_time, "\nSerial Run Time");

for(int i=0; i<length; i++){
     cout << char(b[i]);
}cout << "\nNumber of tries: " << tries << endl;

//long long serial_end_time = stop_timer(serial_start_time, "\nSerial Run Time");

//declare GPU memory pointers
  int *d_output;
//allocate GPU memory
  hipMalloc((void **) &d_output,1000*sizeof(int));
  hipMalloc((void **) &d_input, result);

hipError_t err = hipSuccess;
//transfer the array to the GPU
err = hipMemcpy(d_input, a, result,hipMemcpyHostToDevice);
 if(err != hipSuccess)
  {
    fprintf(stderr, "Failed to copy d_S from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }


//launch the kernel
int threads =length;

long long parallel_start_time = start_timer();

parallel_passwordCrack<<<1,threads>>>(length,d_output,d_input);

long long parallel_end_time = stop_timer(parallel_start_time, "\nParallel Run Time");

//copy back the result array to the CPU

hipMemcpy(h_gpu_result,d_output,1000*sizeof(int),hipMemcpyDeviceToHost);


cout << "\nParallel Password Cracked: " << endl;
for(int i=0; i<length; i++){
	printf("%c\n", char(h_gpu_result[i]));
}
printf("\n");

hipFree(d_output);
hipFree(d_input);
free(h_gpu_result);

return 0;
}
