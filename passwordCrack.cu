
#include <hip/hip_runtime.h>
#include<iostream>
#include<string>
#include<cstring>
#include<ctime>
#include<cstdlib>
#include<sys/time.h>
#include<stdio.h>
#include<iomanip>

using namespace std;

int a[1000]; //array of all possible password characters
int b[1000]; //array of attempted password cracks
unsigned long long tries = 0;
char alphabet[] = { 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z' };


void serial_passwordCrack(int length){
bool cracked = false;
do{
    b[0]++;
    for(int i =0; i<length; i++){
        if (b[i] >= 26 + alphabet[i]){ 
            b[i] -= 26; 
            b[i+1]++;
        }else break;
    }
    cracked=true;
    for(int k=0; k<length; k++)
        if(b[k]!=a[k]){
            cracked=false;
            break;
        }
    if( (tries & 0x7ffffff) == 0 )
        cout << "\r       \r   ";
    else if( (tries & 0x1ffffff) == 0 )
        cout << ".";
    tries++;
}while(cracked==false);

}


__global__ void parallel_passwordCrack(int length)
{

}


long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, std::string name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
        std::cout << std::setprecision(5);	
	std::cout << name << ": " << ((float) (end_time - start_time)) / (1000 * 1000) << " sec\n";
	return end_time - start_time;
}



int main()
{
int length; //length of password
int random; //random password to be generated

cout << "Enter a password length: ";
cin >> length;


srand(time(NULL));
cout << "Random generated password: " << endl;
for (int i =0; i<length; i++){
    
        random = alphabet[(rand()%26)]; 
    
    a[i] = random; //adding random password to array
    cout << char(a[i]);
}cout << "\n" << endl;
cout << "Serial Password Cracked: " << endl;
serial_passwordCrack(length);
cout << "\n";
for(int i=0; i<length; i++){
    cout << char(b[i]);
}cout << "\nNumber of tries: " << tries << endl;

return 0;
}
