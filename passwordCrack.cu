#include "hip/hip_runtime.h"
#include<iostream>
#include<string>
#include<cstring>
#include<ctime>
#include<cstdlib>
#include<sys/time.h>
#include<stdio.h>
#include<iomanip>

using namespace std;

int a[1000]; //array of all possible password characters
int b[1000]; //array of attempted password cracks
unsigned long long tries = 0;
char alphabet[] = { 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z' };
size_t result = 1000 * sizeof(float);

void serial_passwordCrack(int length){
bool cracked = false;
do{
    b[0]++;
    for(int i =0; i<length; i++){
        if (b[i] >= 26 + alphabet[i]){ 
            b[i] -= 26; 
            b[i+1]++;
        }else break;
    }
    cracked=true;
    for(int k=0; k<length; k++)
        if(b[k]!=a[k]){
            cracked=false;
            break;
        }
    if( (tries & 0x7ffffff) == 0 )
        cout << "\r       \r   ";
    else if( (tries & 0x1ffffff) == 0 )
        cout << ".";
    tries++;
}while(cracked==false);

}


__global__ void parallel_passwordCrack(int length,int*d_output,int* a, long attempts)
{	
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	bool cracked = false;
	int mark=0;
        char alphabetTable[] = { 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z' };        
	int newB[1000]; 
 
char alph;	
 
while(!cracked){

	alph = alphabetTable[rand()%26];
	d_output[idx] = int(alph);
	__syncthreads();
	for(int i = 0; i<length; i++){
		if(d_output[i] != a[i])
		{
			cracked = false;
		}
		else{
		cracked = true;
		}

	}

  }
    

}

long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, std::string name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
        std::cout << std::setprecision(5);	
	std::cout << name << ": " << ((float) (end_time - start_time)) / (1000 * 1000) << " sec\n";
	return end_time - start_time;
}



int main()
{
int length; //length of password
int random; //random password to be generated
long attempts; //number of attempts to crack the password
int *d_input = (int *) malloc(result);

cout << "Enter a password length: ";
cin >> length;
int *h_gpu_result = (int*)malloc(1000*sizeof(int));

srand(time(NULL));
cout << "Random generated password: " << endl;
for (int i =0; i<length; i++){
    
        random = alphabet[(rand()%26)]; 
    
    a[i] = random; //adding random password to array
    cout << char(a[i]);
}cout << "\n" << endl;

//declare GPU memory pointers
  int *d_output;
//allocate GPU memory
  hipMalloc((void **) &d_output,1000*sizeof(int));
  hipMalloc((void **) &d_input, result);

hipError_t err = hipSuccess;
//transfer the array to the GP
err = hipMemcpy(d_input, a, result,hipMemcpyHostToDevice);
 if(err != hipSuccess)
  {
    fprintf(stderr, "Failed to copy d_S from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

//launch the kernel
int threads =1;
parallel_passwordCrack<<<1,threads>>>(length,d_output,d_input,attempts);
//copy back the result array to the CPU
hipMemcpy(h_gpu_result,d_output,1000*sizeof(int),hipMemcpyDeviceToHost);

cout << "Serial Password Cracked: " << endl;
serial_passwordCrack(length);
cout << "\n";
for(int i=0; i<length; i++){
    cout << char(b[i]);
}
cout << "\nNumber of tries: " << tries << endl;

cout << "\nParallel Password Cracked: " << endl;
for(int i=0; i<length; i++){
	printf("%c\n", char(h_gpu_result[i]));
}
cout << "\nNumber of attempts: " << attempts << endl;

hipFree(d_output);
hipFree(d_input);
free(h_gpu_result);

return 0;
}
